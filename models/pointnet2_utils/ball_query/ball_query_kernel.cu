#include <stdint.h>
#include <hip/hip_runtime.h>


__global__ void ball_query_kernel(const float* __restrict__ batched_points_xyz,
								  const float* __restrict__ batched_centroids_xyz,
								  int n_points, int n_centroids,
								  float radius, int n_points_per_group,
								  int64_t* __restrict__ batched_indices) {
	int bid = blockIdx.x, tid = threadIdx.x, block_size = blockDim.x;

	const float* __restrict__ points_xyz = batched_points_xyz + bid * n_points * 3;
	const float* __restrict__ centroids_xyz = batched_centroids_xyz + bid * n_centroids * 3;
	int64_t* __restrict__ all_indices = batched_indices + bid * n_centroids * n_points_per_group;

	float radius2 = radius * radius;
	for (int i = tid; i < n_centroids; i += block_size) {
		int64_t* __restrict__ indices = all_indices + i * n_points_per_group;

		float cx = centroids_xyz[i * 3 + 0];
		float cy = centroids_xyz[i * 3 + 1];
		float cz = centroids_xyz[i * 3 + 2];

		int count = 0;
		for (int j = 0; j < n_points && count < n_points_per_group; j++) {
			float px = points_xyz[j * 3 + 0];
			float py = points_xyz[j * 3 + 1];
			float pz = points_xyz[j * 3 + 2];
			float d = (cx - px) * (cx - px) + (cy - py) * (cy - py) + (cz - pz) * (cz - pz);
			if (d < radius2) indices[count++] = j;
		}

		for (int j = count; j < n_points_per_group; j++)
			indices[j] = indices[0];
	}
}

void ball_query_kernel_wrapper(const float* batched_points_xyz, const float* batched_centroids_xyz,
							   int n_batches, int n_points, int n_centroids, float radius, int n_points_per_group,
							   int64_t* batched_indices) {
	ball_query_kernel<<<n_batches, 1024>>>(batched_points_xyz, batched_centroids_xyz, n_points, n_centroids, radius, n_points_per_group, batched_indices);
}